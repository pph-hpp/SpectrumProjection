#include "hip/hip_runtime.h"
#include "../stdafx.h"
#include "ray.cuh"

#define PI 3.1415926536f
#define STEPSIZE 0.2f

__global__ void InitDistance(float* distance_array, const float distance, const int V)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < V)
	{
		distance_array[tid] = distance;
	}
}

__global__ void InitU(float* u, const int N, const float du)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N)
	{
		u[tid] = (tid - (N - 1) / 2.0f) * du;
	}
}

__global__ void InitBeta(float* beta, const int V, const float startAngle, const float totalScanAngle)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < V)
	{
		beta[tid] = (totalScanAngle / V * tid + startAngle) * PI / 180.0f;
	}
}

// img: image data
// sgm: sinogram data
// u: array of each detector element position
// beta: array of each view angle [radian]
// M: image dimension
// S: number of image slices
// N_z: number of detector elements in Z direction
// N: number of detector elements (sinogram width)
// V: number of views (sinogram height)
// dx: image pixel size [mm]
// dz: image slice thickness [mm]
// sid: source to isocenter distance
// sdd: source to detector distance
__global__ void ForwardProjectionBilinear_device(float* img, float* sgm, const float* u, const float* v,\
	const float* beta, int M, int S, int N, int N_z, int V, float dx, float dz, const float sid,\
	const float sdd, bool conebeam, int z_element_begin_idx, int z_element_end_idx)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;


	if (col < N && row < V && z_element_end_idx <= N_z)
	{
		// half of image side length
		float D = M * dx / 2.0f;
		// half of image thickness
		float D_z = 0.0f;
		if (conebeam)
		{
			D_z = float(S) * dz / 2.0f;
		}
		else
		{
			dz = 0;
		}
		//get the sid and sdd for a given view

		// current source position
		float xs = sid * cosf(beta[row]);
		float ys = sid * sinf(beta[row]);
		float zs = 0;

		// current detector element position
		float xd = -(sdd - sid) * cosf(beta[row]) + u[col] * cosf(beta[row] - PI / 2.0f);
		float yd = -(sdd - sid) * sinf(beta[row]) + u[col] * sinf(beta[row] - PI / 2.0f);
		float zd = 0;

		// step point region
		float L_min = sid - sqrt(2 * D * D + D_z * D_z);
		float L_max = sid + sqrt(2 * D * D + D_z * D_z);

		// source to detector element distance
		float sed = sqrtf((xs - xd) * (xs - xd) + (ys - yd) * (ys - yd));// for fan beam case

		// the point position
		float x, y, z;
		// the point index
		int kx, ky, kz;
		// weighting factor for linear interpolation
		float wx, wy, wz;
		float v1, v2;

		// the most upper left image pixel position
		float x0 = -D + dx / 2.0f;
		float y0 = D - dx / 2.0f;
		float z0 = 0;
		if (conebeam)
		{
			z0 = -D_z + dz / 2.0f;// first slice is at the bottom
		}

		// repeat for each slice
		for (int slice = z_element_begin_idx; slice < z_element_end_idx; slice++)
		{
			// initialization
			//sgm[row*N + col + N * V * slice] = 0;
			sgm[row * N + col] = 0;
			if (conebeam)
			{

				zd = v[slice];

				sed = sqrtf((xs - xd) * (xs - xd) + (ys - yd) * (ys - yd) + (zs - zd) * (zs - zd));
			}

			// calculate line integration
			for (float L = L_min; L <= L_max; L += STEPSIZE * sqrt(dx * dx + dz * dz / 2.0f))
			{
				// get the current point position 
				x = xs + (xd - xs) * L / sed;
				y = ys + (yd - ys) * L / sed;
				if (conebeam)
				{
					z = zs + (zd - zs) * L / sed;
				}

				// get the current point index
				kx = floorf((x - x0) / dx);
				ky = floorf((y0 - y) / dx);

				if (conebeam)
					kz = floorf((z - z0) / dz);

				// get the image pixel value at the current point
				if (kx >= 0 && kx + 1 < M && ky >= 0 && ky + 1 < M)
				{
					// get the weighting factor
					/*wx = ((x - x0) - kx * dx) / dx;
					wy = ((y0 - y) - ky * dx) / dx;*/
					wx = (x - kx * dx - x0) / dx;
					wy = (y0 - y - ky * dx) / dx;

					// perform bilinear interpolation
					if (conebeam == false)
					{
						//˫���Բ�ֵ
						/*v1 = (1 - wx) * img[ky * M + kx + M * M * slice]
							+ wx * img[ky * M + kx + 1 + M * M * slice];
						v2 = (1 - wx) * img[(ky + 1) * M + kx + M * M * slice]
							+ wx * img[(ky + 1) * M + kx + 1 + M * M * slice];
						sgm[row * N + col] += (1 - wy) * v1 + wy * v2;*/
						sgm[row * N + col] += (1 - wx) * (1 - wy) * img[ky * M + kx + M * M * slice] // upper left
							+ wx * (1 - wy) * img[ky * M + kx + 1 + M * M * slice] // upper right
							+ (1 - wx) * wy * img[(ky + 1) * M + kx + M * M * slice] // bottom left
							+ wx * wy * img[(ky + 1) * M + kx + 1 + M * M * slice];	// bottom right
						
					}
					else if (conebeam == true && kz >= 0 && kz + 1 < S)
					{
						wz = (z - kz * dz - z0) / dz;
						float sgm_val_lowerslice = (1 - wx) * (1 - wy) * img[ky * M + kx + M * M * kz] // upper left
							+ wx * (1 - wy) * img[ky * M + kx + 1 + M * M * kz] // upper right
							+ (1 - wx) * wy * img[(ky + 1) * M + kx + M * M * kz] // bottom left
							+ wx * wy * img[(ky + 1) * M + kx + 1 + M * M * kz];	// bottom right
						float sgm_val_upperslice = (1 - wx) * (1 - wy) * img[ky * M + kx + M * M * (kz + 1)] // upper left
							+ wx * (1 - wy) * img[ky * M + kx + 1 + M * M * (kz + 1)] // upper right
							+ (1 - wx) * wy * img[(ky + 1) * M + kx + M * M * (kz + 1)] // bottom left
							+ wx * wy * img[(ky + 1) * M + kx + 1 + M * M * (kz + 1)];	// bottom right

						sgm[row * N + col] += (1 - wz) * sgm_val_lowerslice + wz * sgm_val_upperslice;
					}

				}
			}

			sgm[row * N + col] *= STEPSIZE * sqrt(dx * dx + dz * dz);

		}
	}
}

// sgm_large: sinogram data before binning
// sgm: sinogram data after binning
// N: number of detector elements (after binning)
// V: number of views
// S: number of slices
// binSize: bin size
__global__ void BinSinogram(float* sgm_large, float* sgm, int N, int V, int S, int binSize)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < N && row < V)
	{
		// repeat for each slice
		for (int slice = 0; slice < S; slice++)
		{
			// initialization
			sgm[row * N + col + N * V * slice] = 0;

			// sum over each bin
			for (int i = 0; i < binSize; i++)
			{
				sgm[row * N + col + N * V * slice] += sgm_large[row * N * binSize + col * binSize + i + slice * N * binSize * V];
			}
			// take average
			sgm[row * N + col + N * V * slice] /= binSize;
		}
	}
}


//new function with Value member to suit all non uniform parameters

void InitializeU_Agent(float*& u, const int N, const float du)
{
	if (u != nullptr)
		hipFree(u);

	hipMalloc((void**)&u, N * sizeof(float));
	InitU << <(N + 511) / 512, 512 >> > (u, N, du);
}

void InitializeBeta_Agent(float*& beta, const int V, const float startAngle, const float totalScanAngle)
{
	if (beta != nullptr)
		hipFree(beta);

	hipMalloc((void**)&beta, V * sizeof(float));
	InitBeta << < (V + 511) / 512, 512 >> > (beta, V, startAngle, totalScanAngle);
}


void ForwardProjectionBilinear_Agent(float*& image, float*& sinogram, const float sid, const float sdd, \
	const float* u, const float* v, const float* beta, const FPConfig& config, int z_element_idx)
{
	dim3 grid((config.detEltCount + 15) / 16, (config.views + 15) / 16);
	dim3 block(16, 16);

	ForwardProjectionBilinear_device << <grid, block >> > (image, sinogram, u, v, beta, config.imgDim, config.sliceCount, \
		config.detEltCount, config.detZEltCount, config.views, config.pixelSize, config.sliceThickness, sid, sdd, config.coneBeam, z_element_idx, z_element_idx + 1);
	
	hipDeviceSynchronize();
}

void BinSinogram(float*& sinogram_large, float*& sinogram, const FPConfig& config)
{
	dim3 grid((config.detEltCount + 7) / 8, (config.views + 7) / 8);
	dim3 block(8, 8);

	BinSinogram << <grid, block >> > (sinogram_large, sinogram, config.detEltCount, config.views, 1, 1);
	// since the sinogram has only one slice, the z_element count is 1

	hipDeviceSynchronize();
}

void SaveSinogramSlice(const char* filename, float*& sinogram_slice, int z_element_idx, const FPConfig& config)
{
	FILE* fp = nullptr;
	if (z_element_idx == 0)
		fp = fopen(filename, "wb");
	else
		fp = fopen(filename, "ab");

	if (fp == nullptr)
	{
		fprintf(stderr, "Cannot save to file %s!\n", filename);
		exit(4);
	}
	fwrite(sinogram_slice, sizeof(float), config.detEltCount * config.views, fp);
	fclose(fp);
}

void MallocManaged_Agent(float*& p, const int size)
{
	hipMallocManaged((void**)&p, size);
}

void FreeMemory_Agent(float*& p)
{
	hipFree(p);
	p = nullptr;
}
